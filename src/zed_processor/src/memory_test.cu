#include <hip/hip_runtime.h>
#include <sys/mman.h>
#include <stdio.h>
#include <string.h>
#include <errno.h>
#include <chrono>

#include "benchmark/benchmark.h"


int readMemory(uint32_t* mem, int length) {
  int i = 0;
  int sum = 0;
  while (length--) {
    sum += mem[i++];
  }

  return sum;
}

void writeMemory(uint32_t* mem, int length) {
  int i = 0;
  int v = 0;
  while (length--) {
    mem[i++] = v++;
  }
}

const int kItems = 10000000;
const int kMemSize = kItems * sizeof(int);

void fillMemory(uint32_t* p) {
  for (int i = 0; i < kItems; ++i) {
    p[i] = 1;
  }
}

#define check(x) { if ((x) != kItems) state.SkipWithError("Validation fail"); }

static void BM_malloc_read(benchmark::State& state) {
  uint32_t* p = (uint32_t*)malloc(kMemSize);
  fillMemory(p);
  while(state.KeepRunning())
    check(readMemory(p, kItems));
  free(p);

  state.SetBytesProcessed(state.iterations() * kMemSize);
}


static void BM_malloc_write(benchmark::State& state) {
  uint32_t* p = (uint32_t*)malloc(kMemSize);
  while(state.KeepRunning())
    writeMemory(p, kItems);
  free(p);
  
  state.SetBytesProcessed(state.iterations() * kMemSize);
}

static void BM_mmap_read(benchmark::State& state) {
  uint32_t* p = (uint32_t*)mmap(
      0, kMemSize, 
      PROT_READ | PROT_WRITE,
      MAP_PRIVATE | MAP_LOCKED | MAP_ANONYMOUS,
      -1, 0);
  fillMemory(p);

  while(state.KeepRunning())
    check(readMemory(p, kItems));

  munmap(p, kMemSize);
  
  state.SetBytesProcessed(state.iterations() * kMemSize);
}


static void BM_mmap_write(benchmark::State& state) {
  uint32_t* p = (uint32_t*)mmap(
      0, kMemSize, 
      PROT_READ | PROT_WRITE,
      MAP_PRIVATE | MAP_LOCKED | MAP_ANONYMOUS,
      -1, 0);

  while(state.KeepRunning())
    writeMemory(p, kItems);

  munmap(p, kMemSize);
  
  state.SetBytesProcessed(state.iterations() * kMemSize);
}

#define cudaSafeCall(x) if ((x)) { \
  state.SkipWithError(hipGetErrorString(hipGetLastError())); \
  return; \
}

static void BM_cuda_copy_h2d(benchmark::State& state) {
  uint32_t* host_p = (uint32_t*)malloc(kMemSize);
  uint32_t* dev_p;
  cudaSafeCall(hipMalloc(&dev_p, kMemSize));
  fillMemory(host_p);

  while (state.KeepRunning()) {
    hipMemcpy(dev_p, host_p, kMemSize, hipMemcpyHostToDevice);
  }

  hipFree(dev_p);
  free(host_p);
  
  state.SetBytesProcessed(state.iterations() * kMemSize);
}

static void BM_cuda_copy_d2h(benchmark::State& state) {
  uint32_t* host_p = (uint32_t*)malloc(kMemSize);
  uint32_t* dev_p;
  cudaSafeCall(hipMalloc(&dev_p, kMemSize));
  fillMemory(host_p);
  hipMemcpy(dev_p, host_p, kMemSize, hipMemcpyHostToDevice);

  while (state.KeepRunning()) {
    hipMemcpy(host_p, dev_p, kMemSize, hipMemcpyDeviceToHost);
  }

  hipFree(dev_p);
  free(host_p);
  
  state.SetBytesProcessed(state.iterations() * kMemSize);
}

static void BM_cuda_copy_h2d_async(benchmark::State& state) {
  uint32_t* host_p = (uint32_t*)malloc(kMemSize);
  uint32_t* dev_p;
  cudaSafeCall(hipMalloc(&dev_p, kMemSize));
  fillMemory(host_p);

  while (state.KeepRunning()) {
    auto start = std::chrono::high_resolution_clock::now();
    hipMemcpyAsync(dev_p, host_p, kMemSize, hipMemcpyHostToDevice);
    auto end = std::chrono::high_resolution_clock::now();
    hipDeviceSynchronize();
    state.SetIterationTime(
        std::chrono::duration_cast<std::chrono::duration<double>>(end - start).count());
  }

  hipFree(dev_p);
  free(host_p);
}

static void BM_cuda_copy_d2h_async(benchmark::State& state) {
  uint32_t* host_p = (uint32_t*)malloc(kMemSize);
  uint32_t* dev_p;
  cudaSafeCall(hipMalloc(&dev_p, kMemSize));
  fillMemory(host_p);
  hipMemcpy(dev_p, host_p, kMemSize, hipMemcpyHostToDevice);

  while (state.KeepRunning()) {
    auto start = std::chrono::high_resolution_clock::now();
    hipMemcpyAsync(host_p, dev_p, kMemSize, hipMemcpyDeviceToHost);
    auto end = std::chrono::high_resolution_clock::now();
    hipDeviceSynchronize();
    state.SetIterationTime(
        std::chrono::duration_cast<std::chrono::duration<double>>(end - start).count());
  }

  hipFree(dev_p);
  free(host_p);
}



static void BM_cuda_malloc_read(benchmark::State& state) {
  uint32_t* p;
  cudaSafeCall(hipHostMalloc(&p, kMemSize));
  fillMemory(p);

  while(state.KeepRunning())
    check(readMemory(p, kItems));

  hipHostFree(p);

  state.SetBytesProcessed(state.iterations() * kMemSize);
}


static void BM_cuda_malloc_write(benchmark::State& state) {
  uint32_t* p;
  cudaSafeCall(hipHostMalloc(&p, kMemSize));

  while(state.KeepRunning())
    writeMemory(p, kItems);

  hipHostFree(p);
  
  state.SetBytesProcessed(state.iterations() * kMemSize);
}

static void BM_pinned_read(benchmark::State& state) {
  uint32_t* p;
  cudaSafeCall(hipHostAlloc(&p, kMemSize, hipHostMallocDefault));
  fillMemory(p);

  while(state.KeepRunning())
    check(readMemory(p, kItems));

  hipHostFree(p);
  
  state.SetBytesProcessed(state.iterations() * kMemSize);
}


static void BM_pinned_write(benchmark::State& state) {
  uint32_t* p;
  cudaSafeCall(hipHostAlloc(&p, kMemSize, hipHostMallocDefault));

  while(state.KeepRunning())
    writeMemory(p, kItems);

  hipHostFree(p);
  
  state.SetBytesProcessed(state.iterations() * kMemSize);
}


static void BM_mapped_read(benchmark::State& state) {
  uint32_t* p;
  cudaSafeCall(hipHostAlloc(&p, kMemSize, hipHostMallocMapped));
  fillMemory(p);

  while(state.KeepRunning())
    check(readMemory(p, kItems));

  hipHostFree(p);
  
  state.SetBytesProcessed(state.iterations() * kMemSize);
}


static void BM_mapped_write(benchmark::State& state) {
  uint32_t* p;
  cudaSafeCall(hipHostAlloc(&p, kMemSize, hipHostMallocMapped));

  while(state.KeepRunning())
    writeMemory(p, kItems);

  hipHostFree(p);
  
  state.SetBytesProcessed(state.iterations() * kMemSize);
}


static void BM_managed_read(benchmark::State& state) {
  uint32_t* p;
  cudaSafeCall(hipMallocManaged(&p, kMemSize));
  fillMemory(p);

  while(state.KeepRunning())
    check(readMemory(p, kItems));

  hipFree(p);
  
  state.SetBytesProcessed(state.iterations() * kMemSize);
}


static void BM_managed_write(benchmark::State& state) {
  uint32_t* p;
  cudaSafeCall(hipMallocManaged(&p, kMemSize));

  while(state.KeepRunning())
    writeMemory(p, kItems);

  hipFree(p);
  
  state.SetBytesProcessed(state.iterations() * kMemSize);
}

BENCHMARK(BM_malloc_read);
BENCHMARK(BM_malloc_write);
/* BENCHMARK(BM_mmap_read); */
/* BENCHMARK(BM_mmap_write); */
BENCHMARK(BM_cuda_copy_h2d);
BENCHMARK(BM_cuda_copy_d2h);
BENCHMARK(BM_cuda_copy_h2d_async)->UseManualTime();
BENCHMARK(BM_cuda_copy_d2h_async)->UseManualTime();
BENCHMARK(BM_cuda_malloc_read);
BENCHMARK(BM_cuda_malloc_write);
BENCHMARK(BM_pinned_read);
BENCHMARK(BM_pinned_write);
BENCHMARK(BM_mapped_read);
BENCHMARK(BM_mapped_write);
BENCHMARK(BM_managed_read);
BENCHMARK(BM_managed_write);

int main(int argc, char** argv) {
  benchmark::Initialize(&argc, argv);
  benchmark::RunSpecifiedBenchmarks();
  return 0;
}
